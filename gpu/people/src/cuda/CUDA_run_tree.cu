#include "hip/hip_runtime.h"
/** 
 * @authors: Cedric Cagniart, Koen Buys, Anatoly Baksheev
 */

//#include <cutil.h>
#include <pcl/gpu/people/tree.h>
#include <pcl/gpu/utils/safe_call.hpp>
#include <pcl/gpu/utils/timers_cuda.hpp>
#include <stdio.h>

#include "internal.h"

texture<unsigned short, 2, hipReadModeElementType> depthTex;
texture<unsigned short, 2, hipReadModeElementType> maskTex;

using pcl::gpu::people::trees::Node;
using pcl::gpu::people::trees::Label;
using pcl::gpu::people::trees::AttribLocation;

typedef unsigned int uint;

__global__ void KernelCUDA_runTree( const int    W,
                                    const int    H,
                                    const float  f,
                                    const int    treeHeight,
                                    const int    numNodes,
                                    const Node*  nodes,
                                    const Label* leaves,
                                    pcl::device::PtrStep<Label> labels)
{
  uint u = blockIdx.x * blockDim.x + threadIdx.x;
  uint v = blockIdx.y * blockDim.y + threadIdx.y;

  if( u >=W ) return;
  if( v >=H ) return;

  // init
  int    depth = tex2D(depthTex, u,v );
  float  scale = f/float(depth);

  // go down the tree
  int nid = 0;
  for(int nodeDepth=0;nodeDepth<treeHeight;++nodeDepth)
  {
    const Node& node = nodes[nid];
    const AttribLocation& loc = node.loc;
    int d1 = tex2D(depthTex, u+float(loc.du1)*scale, v+float(loc.dv1)*scale);
    int d2 = tex2D(depthTex, u+float(loc.du2)*scale, v+float(loc.dv2)*scale);
    int delta = d1-d2;
    bool test = delta > int(node.thresh);
    if( test ) nid = nid*2+2;
    else       nid = nid*2+1;
  }

  // we try to synchronize the write
  __syncthreads();
  labels.ptr(v)[u] = leaves[nid-numNodes];
}

void pcl::device::CUDA_runTree( const float  focal,
                   const int    treeHeight,
                   const int    numNodes,
                   const void*  nodes_device,
                   const void*  leaves_device,
                   const Depth& depth,
                   Labels& labels )
{
  labels.create( depth.rows(), depth.cols() );

  using pcl::gpu::divUp; 
  pcl::gpu::ScopeTimer scope(__FUNCTION__);  

  int W = depth.cols();
  int H = depth.rows();
  
  hipChannelFormatDesc channeldesc = hipCreateChannelDesc<unsigned short>();

  depthTex.addressMode[0] = hipAddressModeClamp;
  cudaSafeCall( hipBindTexture2D(0, depthTex, depth.ptr(), channeldesc, W, H, depth.step()) );
    

  dim3 block(16, 16);
  dim3 grid( divUp(W, block.x), divUp(H, block.y) );
  
  KernelCUDA_runTree<<< grid, block >>>( W, H, focal, treeHeight, numNodes, 
                                                   (const Node*)  nodes_device, 
                                                   (const Label*) leaves_device, 
                                                   labels);

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
  cudaSafeCall( hipUnbindTexture(depthTex) );        
}



void pcl::device::CUDA_runTree_masked( const float  focal,
                          const int    treeHeight,
                          const int    numNodes,
                          const void*  nodes_device,
                          const void*  leaves_device,
                          const Depth& depth,
                          const void*  mask_in_device,
                          Labels& labels )

{
  labels.create( depth.rows(), depth.cols() );
  int W = depth.cols();
  int H = depth.rows();

  using pcl::gpu::divUp; 
  pcl::gpu::ScopeTimer scope(__FUNCTION__);  

  depthTex.addressMode[0] = hipAddressModeClamp;
  hipChannelFormatDesc channeldesc = hipCreateChannelDesc<unsigned short>();  
  cudaSafeCall( hipBindTexture2D(0, depthTex, depth.ptr(), channeldesc, W, H, depth.step()) );
  cudaSafeCall( hipBindTexture2D(0, maskTex, mask_in_device, channeldesc, W, H, W*sizeof(unsigned short)) );

  dim3 block(16, 16);
  dim3 grid( divUp(W, block.x), divUp(H, block.y) );
  
#if 0
  KernelCUDA_runTree_masked<<< grid, block >>>( W, H, focal, treeHeight, numNodes, 
                                                   (const Node*)  nodes_device, 
                                                   (const Label*) leaves_device, 
                                                   (Label*)       label_out_device);

#endif

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
  
  cudaSafeCall( hipUnbindTexture(depthTex) );
  cudaSafeCall( hipUnbindTexture(maskTex) );      
}

