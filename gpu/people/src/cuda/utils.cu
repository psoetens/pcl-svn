#include "hip/hip_runtime.h"
#include "internal.h"
#include <pcl/gpu/utils/safe_call.hpp>
#include <pcl/gpu/utils/device/limits.hpp>
#include "npp.h"

namespace pcl
{
  namespace device
  {
    __global__ void c2dKernel(const PtrSz<float8> cloud, int cols, PtrStep<unsigned short> depth)
    {
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      if (idx < cloud.size)
      {
        float d = cloud.data[idx].z * 1000; // m -> mm
        d = isnan(d) ? 0 : d;

        int x = idx % cols;
        int y = idx / cols;
        depth.ptr(y)[x] = d;
      }
    }
  }
}

void 
pcl::device::convertCloud2Depth(const DeviceArray<float8>& cloud, int rows, int cols, Depth& depth)
{
  depth.create(rows, cols);

  int block = 256;
  int total = (int)cloud.size();

  c2dKernel<<<divUp(total, block), block>>>(cloud, cols, depth);
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
}

namespace pcl
{
  namespace device
  {    
    texture<uchar4, hipTextureType1D, hipReadModeElementType> cmapTex;

    __global__ void colorKernel(const PtrStepSz<unsigned char> labels, PtrStep<uchar4> rgba)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x < labels.cols && y < labels.rows)
      {
        int l = labels.ptr(y)[x];
        rgba.ptr(y)[x] = tex1Dfetch(cmapTex, l);         
      }
    }
  }
}

void pcl::device::colorLMap(const Labels& labels, const DeviceArray<uchar4>& map, Image& rgba)
{  
  cmapTex.addressMode[0] = hipAddressModeClamp;
  hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();  
  cudaSafeCall( hipBindTexture(0, cmapTex, map.ptr(), desc, map.size() * sizeof(uchar4) ) );
    
  dim3 block(32, 8);
  dim3 grid( divUp(labels.cols(), block.x), divUp(labels.rows(), block.y) );
  
  colorKernel<<< grid, block >>>( labels, rgba );

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
  cudaSafeCall( hipUnbindTexture(cmapTex) );        
}


///////////////////////////////////////////////////////////////////////////////////////////////////////
/// TODO implement getError string for NPP and move this to the same place with cudaSafeCall

#if defined(__GNUC__)
  #define nppSafeCall(expr)  pcl::gpu::___nppSafeCall(expr, __FILE__, __LINE__, __func__)    
#else /* defined(__HIPCC__) || defined(__MSVC__) */
  #define nppSafeCall(expr)  pcl::gpu::___nppSafeCall(expr, __FILE__, __LINE__)    
#endif

namespace pcl
{
  namespace gpu
  {

    void ___nppSafeCall(int err_code, const char *file, const int line, const char *func = "")
    {    
      if (err_code < 0)
      {
          char buf[4096];
          sprintf(buf, "NppErrorCode = %d", err_code);
          error(buf, file, line, func);   
      }    
    }
  }
}


void pcl::device::setZero(Mask& mask)
{
  NppiSize sz = { mask.cols(), mask.rows() };  
  nppSafeCall( nppiSet_8u_C1R( 0, mask.ptr(), (int)mask.step(), sz) );
}

///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
    __global__ void fgDepthKernel(const PtrStepSz<unsigned short> depth1, const PtrStep<unsigned char> inv_mask, PtrStep<unsigned short> depth2)
    {
      int x = blockIdx.x * blockDim.x + threadIdx.x;
      int y = blockIdx.y * blockDim.y + threadIdx.y;

      if (x < depth1.cols && y < depth1.rows)              
      {
        unsigned short d = depth1.ptr(y)[x];
        depth2.ptr(y)[x] = inv_mask.ptr(y)[x] ? d : numeric_limits<unsigned short>::max();         
      }
    }
  }
}

void pcl::device::prepareForeGroundDepth(const Depth& depth1, Mask& inverse_mask, Depth& depth2)
{
  int cols = depth1.cols();
  int rows = depth1.rows();

  depth2.create(rows, cols);
    
  dim3 block(32, 8);
  dim3 grid( divUp(cols, block.x), divUp(rows, block.y) );
  
  fgDepthKernel<<< grid, block >>>( depth1, inverse_mask, depth2 );

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );

}
