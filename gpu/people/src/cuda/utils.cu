#include "hip/hip_runtime.h"
#include "internal.h"
#include <pcl/gpu/utils/safe_call.hpp>
#include <pcl/gpu/utils/device/limits.hpp>
#include "npp.h"

#include <stdio.h>

namespace pcl
{
  namespace device
  {
    __global__ void c2dKernel(const PtrSz<float8> cloud, int cols, PtrStep<unsigned short> depth)
    {
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      if (idx < cloud.size)
      {
        float d = cloud.data[idx].z * 1000; // m -> mm
        d = isnan(d) ? 0 : d;

        int x = idx % cols;
        int y = idx / cols;
        depth.ptr(y)[x] = d;
      }
    }
  }
}

void 
pcl::device::convertCloud2Depth(const DeviceArray<float8>& cloud, int rows, int cols, Depth& depth)
{
  depth.create(rows, cols);

  int block = 256;
  int total = (int)cloud.size();

  c2dKernel<<<divUp(total, block), block>>>(cloud, cols, depth);
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
}

namespace pcl
{
  namespace device
  {
    texture<uchar4, hipTextureType1D, hipReadModeElementType> cmapTex;

    __global__ void colorKernel(const PtrStepSz<unsigned char> labels, PtrStep<uchar4> rgba)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x < labels.cols && y < labels.rows)
      {
        int l = labels.ptr(y)[x];
        rgba.ptr(y)[x] = tex1Dfetch(cmapTex, l);
      }
    }
  }
}

void pcl::device::colorLMap(const Labels& labels, const DeviceArray<uchar4>& map, Image& rgba)
{
  cmapTex.addressMode[0] = hipAddressModeClamp;
  hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();  
  cudaSafeCall( hipBindTexture(0, cmapTex, map.ptr(), desc, map.size() * sizeof(uchar4) ) );

  dim3 block(32, 8);
  dim3 grid( divUp(labels.cols(), block.x), divUp(labels.rows(), block.y) );

  colorKernel<<< grid, block >>>( labels, rgba );

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
  cudaSafeCall( hipUnbindTexture(cmapTex) );
}


///////////////////////////////////////////////////////////////////////////////////////////////////////
/// TODO implement getError string for NPP and move this to the same place with cudaSafeCall

#if defined(__GNUC__)
  #define nppSafeCall(expr)  pcl::gpu::___nppSafeCall(expr, __FILE__, __LINE__, __func__)    
#else /* defined(__HIPCC__) || defined(__MSVC__) */
  #define nppSafeCall(expr)  pcl::gpu::___nppSafeCall(expr, __FILE__, __LINE__)    
#endif

namespace pcl
{
  namespace gpu
  {

    void ___nppSafeCall(int err_code, const char *file, const int line, const char *func = "")
    {
      if (err_code < 0)
      {
          char buf[4096];
          sprintf(buf, "NppErrorCode = %d", err_code);
          error(buf, file, line, func);
      }
    }
  }
}


void pcl::device::setZero(Mask& mask)
{
  NppiSize sz;
  sz.width  = mask.cols();
  sz.height = mask.rows();   
  nppSafeCall( nppiSet_8u_C1R( 0, mask, (int)mask.step(), sz) );
}

void pcl::device::Dilatation::prepareRect5x5Kernel(DeviceArray<unsigned char>& kernel)
{
  if (kernel.size() == KSIZE_X * KSIZE_Y)
    return;

  std::vector<unsigned char> host(KSIZE_X * KSIZE_Y, (unsigned char)1);
  kernel.upload(host);
}

void pcl::device::Dilatation::invoke(const Mask& src, const Kernel& kernel, Mask& dst)
{
  dst.create(src.rows(), src.cols());  

  NppiSize sz;
  sz.width  = src.cols() - KSIZE_X;
  sz.height = src.rows() - KSIZE_Y; 

  NppiSize ksz;
  ksz.width  = KSIZE_X;
  ksz.height = KSIZE_Y;

  NppiPoint anchor;
  anchor.x = ANCH_X;
  anchor.y = ANCH_Y;

  nppSafeCall( nppiDilate_8u_C1R(src.ptr(ANCH_Y) + ANCH_X, (int)src.step(), 
                                 dst.ptr(ANCH_Y) + ANCH_X, (int)dst.step(), sz, kernel, ksz, anchor) );
}

///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
    __global__ void fgDepthKernel(const PtrStepSz<unsigned short> depth1, const PtrStep<unsigned char> inv_mask, PtrStep<unsigned short> depth2)
    {
      int x = blockIdx.x * blockDim.x + threadIdx.x;
      int y = blockIdx.y * blockDim.y + threadIdx.y;

      if (x < depth1.cols && y < depth1.rows)
      {
        unsigned short d = depth1.ptr(y)[x];
        depth2.ptr(y)[x] = inv_mask.ptr(y)[x] ? d : numeric_limits<short>::max();
      }
    }
  }
}

void pcl::device::prepareForeGroundDepth(const Depth& depth1, Mask& inverse_mask, Depth& depth2)
{
  int cols = depth1.cols();
  int rows = depth1.rows();

  depth2.create(rows, cols);

  dim3 block(32, 8);
  dim3 grid( divUp(cols, block.x), divUp(rows, block.y) );

  fgDepthKernel<<< grid, block >>>( depth1, inverse_mask, depth2 );

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
}
