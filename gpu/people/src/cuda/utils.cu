#include "hip/hip_runtime.h"
#include "internal.h"
#include <pcl/gpu/utils/safe_call.hpp>
#include <pcl/gpu/utils/texture_binder.hpp>
#include <pcl/gpu/utils/device/limits.hpp>
#include "npp.h"

#include <stdio.h>

namespace pcl
{
  namespace device
  {
    texture<uchar4, hipTextureType1D, hipReadModeElementType> cmapTex;

    __global__ void colorKernel(const PtrStepSz<unsigned char> labels, PtrStep<uchar4> rgba)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x < labels.cols && y < labels.rows)
      {
        int l = labels.ptr(y)[x];
        rgba.ptr(y)[x] = tex1Dfetch(cmapTex, l);
      }
    }
  }
}

void pcl::device::colorLMap(const Labels& labels, const DeviceArray<uchar4>& map, Image& rgba)
{
  cmapTex.addressMode[0] = hipAddressModeClamp;
  TextureBinder binder(map, cmapTex);
  
  dim3 block(32, 8);
  dim3 grid( divUp(labels.cols(), block.x), divUp(labels.rows(), block.y) );

  colorKernel<<< grid, block >>>( labels, rgba );

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );  
}


///////////////////////////////////////////////////////////////////////////////////////////////////////
/// TODO implement getError string for NPP and move this to the same place with cudaSafeCall

#if defined(__GNUC__)
  #define nppSafeCall(expr)  pcl::gpu::___nppSafeCall(expr, __FILE__, __LINE__, __func__)    
#else /* defined(__HIPCC__) || defined(__MSVC__) */
  #define nppSafeCall(expr)  pcl::gpu::___nppSafeCall(expr, __FILE__, __LINE__)    
#endif

namespace pcl
{
  namespace gpu
  {

    void ___nppSafeCall(int err_code, const char *file, const int line, const char *func = "")
    {
      if (err_code < 0)
      {
          char buf[4096];
          sprintf(buf, "NppErrorCode = %d", err_code);
          error(buf, file, line, func);
      }
    }
  }
}


void pcl::device::setZero(Mask& mask)
{
  NppiSize sz;
  sz.width  = mask.cols();
  sz.height = mask.rows();   
  nppSafeCall( nppiSet_8u_C1R( 0, mask, (int)mask.step(), sz) );
}

void pcl::device::Dilatation::prepareRect5x5Kernel(DeviceArray<unsigned char>& kernel)
{
  if (kernel.size() == KSIZE_X * KSIZE_Y)
    return;

  std::vector<unsigned char> host(KSIZE_X * KSIZE_Y, (unsigned char)255);
  kernel.upload(host);
}

void pcl::device::Dilatation::invoke(const Mask& src, const Kernel& kernel, Mask& dst)
{
  dst.create(src.rows(), src.cols());  
  setZero(dst);

  NppiSize sz;
  sz.width  = src.cols() - KSIZE_X;
  sz.height = src.rows() - KSIZE_Y; 

  NppiSize ksz;
  ksz.width  = KSIZE_X;
  ksz.height = KSIZE_Y;

  NppiPoint anchor;
  anchor.x = ANCH_X;
  anchor.y = ANCH_Y;

  // This one uses Nvidia performance primitives
  nppSafeCall( nppiDilate_8u_C1R(src.ptr(ANCH_Y) + ANCH_X, (int)src.step(), 
                                 dst.ptr(ANCH_Y) + ANCH_X, (int)dst.step(), sz, kernel, ksz, anchor) );
}

///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
    __global__ void fgDepthKernel(const PtrStepSz<unsigned short> depth1, const PtrStep<unsigned char> inv_mask, PtrStep<unsigned short> depth2)
    {
      int x = blockIdx.x * blockDim.x + threadIdx.x;
      int y = blockIdx.y * blockDim.y + threadIdx.y;

      if (x < depth1.cols && y < depth1.rows)
      {
        unsigned short d = depth1.ptr(y)[x];
        depth2.ptr(y)[x] = inv_mask.ptr(y)[x] ? d : numeric_limits<unsigned short>::max();
      }
    }
  }
}

void pcl::device::prepareForeGroundDepth(const Depth& depth1, Mask& inverse_mask, Depth& depth2)
{
  int cols = depth1.cols();
  int rows = depth1.rows();

  depth2.create(rows, cols);

  dim3 block(32, 8);
  dim3 grid( divUp(cols, block.x), divUp(rows, block.y) );

  fgDepthKernel<<< grid, block >>>( depth1, inverse_mask, depth2 );

  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
}
