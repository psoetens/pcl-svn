/*
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include "cuda_interface.hpp"
#include "search/batch_appNearestSearch.hpp"

#include "utils/funcattrib.hpp"

void pcl::gpu::OctreeImpl::approxNearestSearchBatch(const BatchQueries& queries, BatchResult& output) const
{
    typedef OctreeImpl::PointType PointType;
    typedef pcl::device::batch_appnearest_search::Batch<PointType> BatchType;


    BatchType batch;
    batch.indices = indices;
    batch.octree = octreeGlobal;

    batch.queries_num = (int)queries.size();        
    batch.output = output;     

    batch.points = points_sorted;
    batch.points_step = points_sorted.step()/points_sorted.elem_size;
    batch.queries = queries;

    int block = pcl::device::batch_appnearest_search::KernelPolicy::CTA_SIZE;
    int grid = (batch.queries_num + block - 1) / block;    



    cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(pcl::device::batch_appnearest_search::KernelB<PointType>), hipFuncCachePreferL1) );

    pcl::device::batch_appnearest_search::KernelB<<<grid, block>>>(batch);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
}