#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include "pcl/gpu/common/timers_cuda.hpp"
#include "pcl/gpu/common/safe_call.hpp"

#include "cuda_interface.hpp"
#include "utils/boxutils.hpp"

using namespace pcl::gpu;
using namespace pcl::cuda;
using namespace pcl::device;
using namespace std;

namespace pcl
{
    namespace cuda
    {
        namespace getcc
        {
            __global__ void get_cc_kernel(int *data)
            {
                data[threadIdx.x + blockDim.x * blockIdx.x] = threadIdx.x;
            }
        }
    }
}

void  pcl::gpu::OctreeImpl::get_gpu_arch_compiled_for(int& bin, int& ptx)
{
    hipFuncAttributes attrs;
    cudaSafeCall( hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(getcc::get_cc_kernel)) );  
    bin = attrs.binaryVersion;
    ptx = attrs.ptxVersion;
}

void pcl::gpu::OctreeImpl::setCloud(const PointCloud& input_points)
{
    points = input_points;
}


int getBitsNum(int interger)
{
    int count = 0;
    while(interger > 0)
    {
        if (interger & 1)
            ++count;
        interger>>=1;
    }
    return count;
} 

struct OctreeIteratorHost
{        
    const static int MAX_LEVELS_PLUS_ROOT = 11;
    int paths[MAX_LEVELS_PLUS_ROOT];          
    int level;

    OctreeIteratorHost()
    {
        level = 0; // root level
        paths[level] = (0 << 8) + 1;                    
    }

    void gotoNextLevel(int first, int len) 
    {   
        ++level;
        paths[level] = (first << 8) + len;        
    }       

    int operator*() const 
    { 
        return paths[level] >> 8; 
    }        

    void operator++()
    {
        while(level >= 0)
        {
            int data = paths[level];

            if ((data & 0xFF) > 1) // there are another siblings, can goto there
            {                           
                data += (1 << 8) - 1;  // +1 to first and -1 from len
                paths[level] = data;
                break;
            }
            else
                --level; //goto parent;            
       }        
    }        
};

void pcl::gpu::OctreeImpl::radiusSearchHost(const PointType& query, float radius, vector<int>& out, int max_nn) const
{            
    out.clear();  

    float3 center = make_float3(query.x, query.y, query.z);

    OctreeIteratorHost iterator;

    while(iterator.level >= 0)
    {        
        int node_idx = *iterator;
        int code = host_octree.node_codes[node_idx];

        float3 node_minp = octreeGlobal.minp;
        float3 node_maxp = octreeGlobal.maxp;        
        calcBoundingBox(iterator.level, code, node_minp, node_maxp);

        //if true, take nothing, and go to next
        if (checkIfNodeOutsideSphere(node_minp, node_maxp, center, radius))        
        {                
            ++iterator;            
            continue;
        }

        //if true, take all, and go to next
        if (checkIfNodeInsideSphere(node_minp, node_maxp, center, radius))
        {            
            int beg = host_octree.begs[node_idx];
            int end = host_octree.ends[node_idx];

            end = beg + min<int>(out.size() + end - beg, max_nn) - out.size();

            out.insert(out.end(), host_octree.indices.begin() + beg, host_octree.indices.begin() + end);
            if (out.size() == max_nn)
                return;

            ++iterator;
            continue;
        }

        // test children
        int children_mask = host_octree.nodes[node_idx] & 0xFF;
        
        bool isLeaf = children_mask == 0;

        if (isLeaf)
        {            
            const int beg = host_octree.begs[node_idx];
            const int end = host_octree.ends[node_idx];                                    

            for(int j = beg; j < end; ++j)
            {
                int index = host_octree.indices[j];
                const float& point_x = host_octree.points_sorted[j                                     ];
                const float& point_y = host_octree.points_sorted[j + host_octree.points_sorted_step    ];
                const float& point_z = host_octree.points_sorted[j + host_octree.points_sorted_step * 2];

                float dx = (point_x - center.x);
                float dy = (point_y - center.y);
                float dz = (point_z - center.z);

                float dist2 = dx * dx + dy * dy + dz * dz;

                if (dist2 < radius * radius)
                    out.push_back(index);
                
                if (out.size() == max_nn)
                    return;
            }               
            ++iterator;               
            continue;
        }
        
        int first  = host_octree.nodes[node_idx] >> 8;
        iterator.gotoNextLevel(first, getBitsNum(children_mask));                
    }
}

void pcl::gpu::OctreeImpl::internalDownload()
{
    int number;
    DeviceArray_<int>(octreeGlobal.nodes_num, 1).download(&number); 

    DeviceArray_<int>(octreeGlobal.begs,  number).download(host_octree.begs);    
    DeviceArray_<int>(octreeGlobal.ends,  number).download(host_octree.ends);    
    DeviceArray_<int>(octreeGlobal.nodes, number).download(host_octree.nodes);    
    DeviceArray_<int>(octreeGlobal.codes, number).download(host_octree.node_codes); 

    points_sorted.download(host_octree.points_sorted, host_octree.points_sorted_step);    
    indices.download(host_octree.indices);    

    host_octree.downloaded = true;
}